#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "defs.h"

#define MP_COUNT                       = 20
#define CORES_PER_MP                   = 128
#define MAX_THREADS_PER_BLOCK          = 1024
#define MAX_THREADS_PER_MULTIPROCESSOR = 2048

// #define THREADS_PER_BLOCK_TARGET 128
// #define BLOCK_TARGET             1024

using namespace std;

float lowerBound;
float upperBound;
float pointIncrement;
int   points;
int   pointsPerThread;
int   blocks;
int   threads;

int threadTarget;
int blockTarget;

float * d_Parameters; // Array where the parameters are copied before a call.
float * d_FitData;    // The fit data to compute RMSE from.
float * d_Sums;       // The array where partial sums are stored for error
                      // calculation. The CPU will take the square root if the
                      // sum of this array before returing,

float * h_Sums; // Host copy of the sums.

__device__ float fitFunction(float * p, float x) {
	return p[0]  * -powf(x - 1.0/1.0,  2.0) + 
		   p[1]  *  powf(x - 1.0/2.0,  2.0) + 
		   p[2]  * -powf(x - 1.0/3.0,  2.0) + 
		   p[3]  *  powf(x - 1.0/4.0,  3.0) + 
		   p[4]  * -powf(x - 1.0/5.0,  2.0) + 
		   p[5]  *  powf(x - 1.0/6.0,  2.0) + 
		   p[6]  * -powf(x - 1.0/7.0,  2.0) + 
		   p[7]  *  powf(x - 1.0/8.0,  3.0) + 
		   p[8]  * -powf(x - 1.0/9.0,  2.0) + 
		   p[9]  *  powf(x - 1.0/10.0, 2.0) + 
		   p[10] * -powf(x - 1.0/11.0, 2.0) + 
		   p[11] *  powf(x - 1.0/12.0, 3.0) + 
		   p[12] * -powf(x - 1.0/13.0, 2.0) + 
		   p[13] *  powf(x - 1.0/14.0, 2.0) + 
		   p[14] * -powf(x - 1.0/15.0, 2.0); 
}

__global__ void computeRMSE(
		float * params,         // Pointer to fit parameters.
		float * fit,            // Pointer to data to fit.
		float * sums,           // Pointer to array to put sums into.
		float   lowerBound,     // Lower bound of the domain.
		float   pointIncrement, // The distance between points in the domain.
		int     pointsPerThread // Number of data points each thread should process.
	) {

	float threadLowerBound =  (blockDim.x * blockIdx.x + threadIdx.x) * 
	                          pointIncrement * pointsPerThread;
	      threadLowerBound += lowerBound;

	int fitIndex = (blockDim.x * blockIdx.x + threadIdx.x) * pointsPerThread;

	for (int i = 0; i < pointsPerThread; ++i) {
		float index_X = threadLowerBound + i * pointIncrement;
		float val     = fitFunction(params, index_X) - fit[fitIndex + i];
		float result  = val * val;

		float * sumAddr = &sums[threadIdx.x];

		atomicAdd(sumAddr, result);
	}
}


extern "C" void configure(float lb, float ub, int p, float * data, int thr, int bl) {
	lowerBound = lb;
	upperBound = ub;
	points     = p;

	pointIncrement = (float)(((double)ub - (double)lb) / (double)p);

	hipMalloc(&d_Parameters, sizeof(float) * PARAMETER_COUNT);
	hipMalloc(&d_FitData,    sizeof(float) * points);
	hipMalloc(&d_Sums,       sizeof(float) * thr);

	h_Sums = (float *)malloc(sizeof(float) * thr);

	hipMemcpy(d_FitData, data, sizeof(float) * points, hipMemcpyHostToDevice);

	// Here we determine the number of blocks and the
	// number of data points to process per thread. These
	// values are determined based on the size of the data
	// and the desired number of threads per block (threads).

	blocks  = bl;
	threads = thr;

	double ppt     = (double)points / (blocks * threads);
	while (ppt != floor(ppt)) {
		ppt = (double)points / (++blocks * threads);
	}

	pointsPerThread = floor(ppt);

	printf("CUDA Parameters:\n");
	printf("Blocks: %d\nThreads/Block: %d\nPoints/Thread: %d\nData Points: %d\n\n", 
		blocks, threads, pointsPerThread, points);
}

extern "C" void finish() {
	hipFree(d_Parameters);
	hipFree(d_FitData);
	hipFree(d_Sums);
	free(h_Sums);
}

extern "C" float getRMSE(float * parameters) {
	hipMemcpy(
		d_Parameters, parameters, sizeof(float) * PARAMETER_COUNT, hipMemcpyHostToDevice);
	hipMemset(d_Sums, 0, sizeof(float) * threads);

	computeRMSE<<<blocks, threads>>>(
		d_Parameters,
		d_FitData,
		d_Sums,
		lowerBound,
		pointIncrement,
		pointsPerThread
	);

	hipMemcpy(
		h_Sums, d_Sums, sizeof(float) * threads, hipMemcpyDeviceToHost);


	float sum = 0.0;
	for (int i = 0; i < threads; ++i) {
		sum += h_Sums[i];
	}

	return sqrtf(sum / (float)points);
}