#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "defs.h"

#define MP_COUNT                       = 20
#define CORES_PER_MP                   = 128
#define MAX_THREADS_PER_BLOCK          = 1024
#define MAX_THREADS_PER_MULTIPROCESSOR = 2048

#define THREADS_PER_BLOCK_TARGET 128
#define BLOCK_TARGET             160

using namespace std;

float lowerBound;
float upperBound;
float pointIncrement;
int   points;
int   pointsPerThread;
int   blocks;
int   threads;

float * d_Parameters; // Array where the parameters are copied before a call.
float * d_FitData;    // The fit data to compute RMSE from.
float * d_Sums;       // The array where partial sums are stored for error
                      // calculation. The CPU will take the square root if the
                      // sum of this array before returing,

float * h_Sums; // Host copy of the sums.

__global__ void computeRMSE(
		float * params,         // Pointer to fit parameters.
		float * fit,            // Pointer to data to fit.
		float * sums,           // Pointer to array to put sums into.
		float   lowerBound,     // Lower bound of the domain.
		float   pointIncrement, // The distance between points in the domain.
		int     pointsPerThread // Number of data points each thread should process.
	) {

	float threadLowerBound =  (blockDim.x * blockIdx.x + threadIdx.x) * 
	                          pointIncrement * pointsPerThread;
	      threadLowerBound += lowerBound;

	int fitIndex = (blockDim.x * blockIdx.x + threadIdx.x) * pointsPerThread;

	for (int i = 0; i < pointsPerThread; ++i) {
		float index_X = threadLowerBound + i * pointIncrement;
		float val     = fitFunction(params, index_X) - fit[fitIndex + i];
		float result  = val * val;

		float * sumAddr = &sums[threadIdx.x];

		atomicAdd(sumAddr, result);
	}
}

__device__ float fitFunction(float * parameters, float x) {
	return p[0]  * -powf(x - 1.0/1.0,  2.0) + 
		   p[1]  *  powf(x - 1.0/2.0,  2.0) + 
		   p[2]  * -powf(x - 1.0/3.0,  2.0) + 
		   p[3]  *  powf(x - 1.0/4.0,  3.0) + 
		   p[4]  * -powf(x - 1.0/5.0,  2.0) + 
		   p[5]  *  powf(x - 1.0/6.0,  2.0) + 
		   p[6]  * -powf(x - 1.0/7.0,  2.0) + 
		   p[7]  *  powf(x - 1.0/8.0,  3.0) + 
		   p[8]  * -powf(x - 1.0/9.0,  2.0) + 
		   p[9]  *  powf(x - 1.0/10.0, 2.0) + 
		   p[10] * -powf(x - 1.0/11.0, 2.0) + 
		   p[11] *  powf(x - 1.0/12.0, 3.0) + 
		   p[12] * -powf(x - 1.0/13.0, 2.0) + 
		   p[13] *  powf(x - 1.0/14.0, 2.0) + 
		   p[14] * -powf(x - 1.0/15.0, 2.0); 
}


void configure(float lb, float ub, int p, float * data) {
	lowerBound = lb;
	upperBound = ub;
	points     = p;

	pointIncrement = (float)(((double)ub - (double)lb) / (double)p);

	hipMalloc(&d_Parameters, sizeof(float) * PARAMETER_COUNT);
	hipMalloc(&d_FitData,    sizeof(float) * points);
	hipMalloc(&d_Sums,       sizeof(float) * THREADS_PER_BLOCK_TARGET);

	h_Sums = malloc(sizeof(float) * THREADS_PER_BLOCK_TARGET);

	hipMemcpy(d_FitData, data, sizeof(float) * points, hipMemcpyHostToDevice);

	// Here we determine the number of blocks and the
	// number of data points to process per thread. These
	// values are determined based on the size of the data
	// and the desired number of threads per block (THREADS_PER_BLOCK_TARGET).

	blocks  = BLOCK_TARGET;
	threads = THREADS_PER_BLOCK_TARGET;

	double ppt     = (double)points / (blocks * threads);
	while (ppt != floor(ppt)) {
		ppt = (double)points / (++blocks * threads);
	}

	pointsPerThread = floor(ppt);

	printf("Blocks: %d, Threads: %d, Points/Thread: %d\n", 
		blocks, threads, pointsPerThread);
}

void finish() {
	hipFree(d_Parameters);
	hipFree(d_FitData);
	hipFree(d_Sums);
	free(h_Sums);
}

float getRMSE(float * parameters) {
	hipMemcpy(d_Parameters, parameters, sizeof(float) * points, hipMemcpyHostToDevice);
	hipMemset(d_Sums, 0, sizeof(float) * THREADS_PER_BLOCK_TARGET);

	computeRMSE<<<blocks, threads>>>(
		d_Parameters,
		d_FitData,
		d_Sums,
		lowerBound,
		pointIncrement,
		pointsPerThread
	);

	hipMemcpy(
		h_Sums, d_Sums, sizeof(float) * THREADS_PER_BLOCK_TARGET, hipMemcpyDeviceToHost);


	float sum = 0.0;
	for (int i = 0; i < THREADS_PER_BLOCK_TARGET; ++i) {
		sum += h_Sums[i];
	}

	return sqrtf(sum);
}