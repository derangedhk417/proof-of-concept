#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "defs.h"

#define MP_COUNT                       = 20
#define CORES_PER_MP                   = 128
#define MAX_THREADS_PER_BLOCK          = 1024
#define MAX_THREADS_PER_MULTIPROCESSOR = 2048

// These values are all set in the command
// line invocation of the compiler.
// #define POINTS_PER_THREAD
// #define BLOCKS
// #define THREADS_PER_BLOCK
// #define POINT_INCREMENT

#define USE_LARGE_ARRAY
#undef USE_LARGE_ARRAY

#define USE_SMALL_ARRAY
#undef USE_SMALL_ARRAY

#define REDUCE_PER_BLOCK
#undef REDUCE_PER_BLOCK
#define SINGLE_THREAD_REDUCE
#undef SINGLE_THREAD_REDUCE

using namespace std;

float lowerBound;
float upperBound;
float pointIncrement;
int   points;
int   pointsPerThread;
int   blocks;
int   threads;

int threadTarget;
int blockTarget;

float * d_Parameters; // Array where the parameters are copied before a call.
float * d_FitData;    // The fit data to compute RMSE from.
float * d_Sums;       // The array where partial sums are stored for error
                      // calculation. The CPU will take the square root if the
                      // sum of this array before returing,

float * h_Sums; // Host copy of the sums.

__device__ float fitFunction(float * p, float x) {
	return p[0]  * -powf(x - 1.0/1.0,  2.0) + 
		   p[1]  *  powf(x - 1.0/2.0,  2.0) + 
		   p[2]  * -powf(x - 1.0/3.0,  2.0) + 
		   p[3]  *  powf(x - 1.0/4.0,  3.0) + 
		   p[4]  * -powf(x - 1.0/5.0,  2.0) + 
		   p[5]  *  powf(x - 1.0/6.0,  2.0) + 
		   p[6]  * -powf(x - 1.0/7.0,  2.0) + 
		   p[7]  *  powf(x - 1.0/8.0,  3.0) + 
		   p[8]  * -powf(x - 1.0/9.0,  2.0) + 
		   p[9]  *  powf(x - 1.0/10.0, 2.0) + 
		   p[10] * -powf(x - 1.0/11.0, 2.0) + 
		   p[11] *  powf(x - 1.0/12.0, 3.0) + 
		   p[12] * -powf(x - 1.0/13.0, 2.0) + 
		   p[13] *  powf(x - 1.0/14.0, 2.0) + 
		   p[14] * -powf(x - 1.0/15.0, 2.0); 
}

#ifdef REDUCE_PER_BLOCK
__shared__ float reduction[THREADS_PER_BLOCK];
#endif

template <int PPT> __global__ void computeRMSE(
		float * params,         // Pointer to fit parameters.
		float * fit,            // Pointer to data to fit.
		float * sums,           // Pointer to array to put sums into.
		float   lowerBound      // Lower bound of the domain.
	) {

#ifdef REDUCE_PER_BLOCK
	reduction[threadIdx.x] = 0.0;
#endif

	float blockLowerBound  = (THREADS_PER_BLOCK * blockIdx.x) * POINT_INCREMENT * PPT;
	float threadStartPoint = lowerBound + blockLowerBound + (threadIdx.x * POINT_INCREMENT);
	int   threadStartIdx   = (THREADS_PER_BLOCK * blockIdx.x * PPT) + threadIdx.x;

#pragma unroll
	for (int i = 0; i < PPT; ++i) {
		float index_X = threadStartPoint + i * POINT_INCREMENT * THREADS_PER_BLOCK;
		float val     = 
			fitFunction(params, index_X) - fit[threadStartIdx + (i * THREADS_PER_BLOCK)];
		float result  = val * val;

	#ifdef USE_LARGE_ARRAY
		sums[threadStartIdx + (i * THREADS_PER_BLOCK)] = result;
	#else
		#ifdef REDUCE_PER_BLOCK
			reduction[threadIdx.x] += result;
		#else
			#ifdef USE_SMALL_ARRAY
				float * sumAddr = &sums[(threadIdx.x % 128)];
			#else
				float * sumAddr = &sums[threadIdx.x];
			#endif
			
			atomicAdd(sumAddr, result);
		#endif
		
		
	#endif
		
	}

	#ifdef REDUCE_PER_BLOCK
		#ifdef SINGLE_THREAD_REDUCE
			if (threadIdx.x == 0) {
				float sum = 0.0;
				#pragma unroll
				for (int i = 0; i < THREADS_PER_BLOCK; ++i) {
					sum += reduction[i];
				}

				sums[blockIdx.x] = sum;
			}
		#else
			atomicAdd(&sums[blockIdx.x], reduction[threadIdx.x]);
		#endif
	#endif
}


extern "C" void configure(float lb, float ub, int p, float * data, int thr, int bl) {
	#ifdef USE_LARGE_ARRAY
		printf("NOTE: File was compiled with USE_LARGE_ARRAY on.\n");
	#endif

	#ifdef USE_SMALL_ARRAY
		printf("NOTE: File was compiled with USE_SMALL_ARRAY on.\n");
	#endif

	lowerBound = lb;
	upperBound = ub;
	points     = p;

	pointIncrement = (float)(((double)ub - (double)lb) / (double)p);

	hipMalloc(&d_Parameters, sizeof(float) * PARAMETER_COUNT);
	hipMalloc(&d_FitData,    sizeof(float) * points);

	#ifdef USE_LARGE_ARRAY
		hipMalloc(&d_Sums, sizeof(float) * points);
		h_Sums = (float *)malloc(sizeof(float) * points);
	#else
		#ifdef REDUCE_PER_BLOCK
			hipMalloc(&d_Sums, sizeof(float) * bl);
			h_Sums = (float *)malloc(sizeof(float) * bl);
		#else
			#ifdef USE_SMALL_ARRAY
				hipMalloc(&d_Sums, sizeof(float) * (thr / 8));
				h_Sums = (float *)malloc(sizeof(float) * (thr / 8));
			#else
				hipMalloc(&d_Sums, sizeof(float) * thr);
				h_Sums = (float *)malloc(sizeof(float) * thr);
			#endif
		#endif
	#endif

	hipMemcpy(d_FitData, data, sizeof(float) * points, hipMemcpyHostToDevice);

	// Here we determine the number of blocks and the
	// number of data points to process per thread. These
	// values are determined based on the size of the data
	// and the desired number of threads per block (threads).

	// blocks  = bl;
	// threads = thr;

	// double ppt     = (double)points / (blocks * threads);
	// while (ppt != floor(ppt)) {
	// 	ppt = (double)points / (++blocks * threads);
	// }

	// pointsPerThread = floor(ppt);

	// printf("CUDA Parameters:\n");
	// printf("Blocks: %d\nThreads/Block: %d\nPoints/Thread: %d\nData Points: %d\n\n", 
	// 	BLOCKS, THREADS_PER_BLOCK, POINTS_PER_THREAD, points);
}

extern "C" void finish() {
	hipFree(d_Parameters);
	hipFree(d_FitData);
	hipFree(d_Sums);
	free(h_Sums);
}

extern "C" float getRMSE(float * parameters) {
	hipMemcpy(
		d_Parameters, parameters, sizeof(float) * PARAMETER_COUNT, hipMemcpyHostToDevice);

	#ifdef USE_LARGE_ARRAY
		hipMemset(d_Sums, 0, sizeof(float) * points);
	#else
		#ifdef REDUCE_PER_BLOCK
			#ifndef SINGLE_THREAD_REDUCE
				// If we are using a single thread at the end to 
				// complete the reduction then we don't need to zero
				// this.
				hipMemset(d_Sums, 0, sizeof(float) * BLOCKS);
			#endif
		#else
			#ifdef USE_SMALL_ARRAY
				hipMemset(d_Sums, 0, sizeof(float) * (THREADS_PER_BLOCK / 8));
			#else
				hipMemset(d_Sums, 0, sizeof(float) * THREADS_PER_BLOCK);
			#endif
		#endif
	#endif
	

	computeRMSE<POINTS_PER_THREAD> <<<BLOCKS, THREADS_PER_BLOCK>>>(
		d_Parameters,
		d_FitData,
		d_Sums,
		lowerBound
	);

	#ifdef USE_LARGE_ARRAY
		hipMemcpy(
			h_Sums, d_Sums, sizeof(float) * points, hipMemcpyDeviceToHost);
	#else
		#ifdef REDUCE_PER_BLOCK
			hipMemcpy(
					h_Sums, d_Sums, sizeof(float) * BLOCKS, hipMemcpyDeviceToHost);
		#else
			#ifdef USE_SMALL_ARRAY
				hipMemcpy(
					h_Sums, d_Sums, sizeof(float) * (THREADS_PER_BLOCK / 8), hipMemcpyDeviceToHost);
			#else
				hipMemcpy(
					h_Sums, d_Sums, sizeof(float) * THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
			#endif
		#endif
	#endif

	


	float sum = 0.0;
#ifdef USE_LARGE_ARRAY
	for (int i = 0; i < points; ++i) {
#else
	#ifdef REDUCE_PER_BLOCK
		for (int i = 0; i < BLOCKS; ++i) {
	#else
		#ifdef USE_SMALL_ARRAY
			for (int i = 0; i < (THREADS_PER_BLOCK / 8); ++i) {
		#else
			for (int i = 0; i < THREADS_PER_BLOCK; ++i) {
		#endif
	#endif
	
#endif
		sum += h_Sums[i];
	}

	return sqrtf(sum / (float)points);
}