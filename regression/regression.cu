#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "defs.h"

#define MP_COUNT                       = 20
#define CORES_PER_MP                   = 128
#define MAX_THREADS_PER_BLOCK          = 1024
#define MAX_THREADS_PER_MULTIPROCESSOR = 2048

using namespace std;

double lowerBound;
double upperBound;
int    points;

double * devParameters;
double * 

void configure(double lb, double ub, int p) {
	lowerBound = lb;
	upperBound = ub;
	points     = p;

	double * devParameters;
	hipMallocManaged(&devParameters, sizeof(double) * PARAMETER_COUNT);
}

void finish() {
	hipFree(devParameters);
}

double getRMSE(double * parameters) {
	for (int i = 0; i < PARAMETER_COUNT; ++i) {
		devParameters[i] = parameters[i];
	}

	hipDeviceSynchronize();


}